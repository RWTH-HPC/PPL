#include "hip/hip_runtime.h"
/**


This file implements the header for the Thread Pool and barrier implementation with PThreads.


*/
#include <stdio.h>
#include <stdlib.h>
#include "cuda_lib_DPNestedTest.hxx"
#include "cuda_lib_DPNestedTest.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "Patternlib.hxx"




template<typename T>
__global__
void cuda_reduce_sum(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] += sdata[tid + 32];
        if (n >= 32) sdata[tid] += sdata[tid + 16];
        if (n >= 16) sdata[tid] += sdata[tid + 8];
        if (n >= 8) sdata[tid] += sdata[tid + 4];
        if (n >= 4) sdata[tid] += sdata[tid + 2];
        if (n >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_times(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] *= sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] *= sdata[tid + 32];
        if (n >= 32) sdata[tid] *= sdata[tid + 16];
        if (n >= 16) sdata[tid] *= sdata[tid + 8];
        if (n >= 8) sdata[tid] *= sdata[tid + 4];
        if (n >= 4) sdata[tid] *= sdata[tid + 2];
        if (n >= 2) sdata[tid] *= sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_min(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_max(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = max(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = max(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = max(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = max(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = max(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = max(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

void cuda_wrapper_matrixIncrement_eqctmEcrHD(int32_t* initial_O7UVdy8jFf, int32_t* result_D7HcKyZsyA ) {

	kernel_cuda_wrapper_matrixIncrement_eqctmEcrHD<<<1, 168>>> (initial_O7UVdy8jFf, result_D7HcKyZsyA );
}

__global__ 
void kernel_cuda_wrapper_matrixIncrement_eqctmEcrHD(int32_t*  initial_O7UVdy8jFf, int32_t* result_D7HcKyZsyA ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 1;
	if (tid < 32) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_eqctmEcrHD = tid * exec_range + range_iterator + 32;
		if (tid < 32) {
			INDEX_eqctmEcrHD -= 32;
		}
		for (size_t INDEX0_aHoGI4RHO3 = 0; INDEX0_aHoGI4RHO3 < 0 + 10; ++INDEX0_aHoGI4RHO3) {
			for (size_t INDEX1_aHoGI4RHO3 = 0; INDEX1_aHoGI4RHO3 < 0 + 200; ++INDEX1_aHoGI4RHO3) {
				result_D7HcKyZsyA[200LL * (INDEX_eqctmEcrHD) + (INDEX1_aHoGI4RHO3)] = initial_O7UVdy8jFf[200LL * (INDEX_eqctmEcrHD) + (INDEX1_aHoGI4RHO3)] + 1;
			}
			Set_Partial_Array( &initial_O7UVdy8jFf[200LL * (INDEX_eqctmEcrHD)], &result_D7HcKyZsyA[200LL * (INDEX_eqctmEcrHD)], 200);
		}
	}
}


