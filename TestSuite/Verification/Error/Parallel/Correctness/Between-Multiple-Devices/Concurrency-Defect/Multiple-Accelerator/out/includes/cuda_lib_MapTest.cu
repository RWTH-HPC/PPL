#include "hip/hip_runtime.h"
/**


This file implements the header for the Thread Pool and barrier implementation with PThreads.


*/
#include <stdio.h>
#include <stdlib.h>
#include "cuda_lib_MapTest.hxx"
#include "cuda_lib_MapTest.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "Patternlib.hxx"




template<typename T>
__global__
void cuda_reduce_sum(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] += sdata[tid + 32];
        if (n >= 32) sdata[tid] += sdata[tid + 16];
        if (n >= 16) sdata[tid] += sdata[tid + 8];
        if (n >= 8) sdata[tid] += sdata[tid + 4];
        if (n >= 4) sdata[tid] += sdata[tid + 2];
        if (n >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_times(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] *= sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] *= sdata[tid + 32];
        if (n >= 32) sdata[tid] *= sdata[tid + 16];
        if (n >= 16) sdata[tid] *= sdata[tid + 8];
        if (n >= 8) sdata[tid] *= sdata[tid + 4];
        if (n >= 4) sdata[tid] *= sdata[tid + 2];
        if (n >= 2) sdata[tid] *= sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_min(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_max(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = max(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = max(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = max(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = max(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = max(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = max(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

void cuda_wrapper_increment_PJDc73oaB9(int32_t* initial_VRUERJiOcV, int32_t* result_jvmvgdiKj9 ) {

	kernel_cuda_wrapper_increment_PJDc73oaB9<<<1, 168>>> (initial_VRUERJiOcV, result_jvmvgdiKj9 );
}

__global__ 
void kernel_cuda_wrapper_increment_PJDc73oaB9(int32_t*  initial_VRUERJiOcV, int32_t* result_jvmvgdiKj9 ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 0;
	if (tid < 100) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_PJDc73oaB9 = tid * exec_range + range_iterator + 100;
		if (tid < 100) {
			INDEX_PJDc73oaB9 -= 100;
		}
		result_jvmvgdiKj9[(INDEX_PJDc73oaB9)] = initial_VRUERJiOcV[(INDEX_PJDc73oaB9)] + 1;
	}
}

void cuda_wrapper_increment_nq9cxVBaee(int32_t* initial_VRUERJiOcV, int32_t* result_jvmvgdiKj9 ) {

	kernel_cuda_wrapper_increment_nq9cxVBaee<<<1, 168>>> (initial_VRUERJiOcV, result_jvmvgdiKj9 );
}

__global__ 
void kernel_cuda_wrapper_increment_nq9cxVBaee(int32_t*  initial_VRUERJiOcV, int32_t* result_jvmvgdiKj9 ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 0;
	if (tid < 100) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_nq9cxVBaee = tid * exec_range + range_iterator + 100;
		if (tid < 100) {
			INDEX_nq9cxVBaee -= 100;
		}
		result_jvmvgdiKj9[(INDEX_nq9cxVBaee)] = initial_VRUERJiOcV[(INDEX_nq9cxVBaee)] + 1;
	}
}


