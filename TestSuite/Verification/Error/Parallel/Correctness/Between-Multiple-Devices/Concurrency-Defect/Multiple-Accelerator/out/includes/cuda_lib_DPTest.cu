#include "hip/hip_runtime.h"
/**


This file implements the header for the Thread Pool and barrier implementation with PThreads.


*/
#include <stdio.h>
#include <stdlib.h>
#include "cuda_lib_DPTest.hxx"
#include "cuda_lib_DPTest.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "Patternlib.hxx"




template<typename T>
__global__
void cuda_reduce_sum(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] += sdata[tid + 32];
        if (n >= 32) sdata[tid] += sdata[tid + 16];
        if (n >= 16) sdata[tid] += sdata[tid + 8];
        if (n >= 8) sdata[tid] += sdata[tid + 4];
        if (n >= 4) sdata[tid] += sdata[tid + 2];
        if (n >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_times(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] *= sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] *= sdata[tid + 32];
        if (n >= 32) sdata[tid] *= sdata[tid + 16];
        if (n >= 16) sdata[tid] *= sdata[tid + 8];
        if (n >= 8) sdata[tid] *= sdata[tid + 4];
        if (n >= 4) sdata[tid] *= sdata[tid + 2];
        if (n >= 2) sdata[tid] *= sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_min(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_max(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = max(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = max(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = max(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = max(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = max(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = max(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

void cuda_wrapper_incr_jCr8Yz9sSg(int32_t* initial_jxAAR4OqHM, int32_t* result_25ndUhg8ZL, int INDEX0_habLccSPNH) {

	kernel_cuda_wrapper_incr_jCr8Yz9sSg<<<1, 168>>> (initial_jxAAR4OqHM, result_25ndUhg8ZL, INDEX0_habLccSPNH);
}

__global__ 
void kernel_cuda_wrapper_incr_jCr8Yz9sSg(int32_t* initial_jxAAR4OqHM, int32_t* result_25ndUhg8ZL, int INDEX0_habLccSPNH) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 0;
	if (tid < 100) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX1_habLccSPNH = tid * exec_range + range_iterator + 100;
		if (tid < 100) {
			INDEX1_habLccSPNH -= 100;
		}
		result_25ndUhg8ZL[(INDEX1_habLccSPNH)] = initial_jxAAR4OqHM[(INDEX1_habLccSPNH)] + 1;
	}
}

void cuda_wrapper_incr_hVGVGWenmY(int32_t* initial_jxAAR4OqHM, int32_t* result_25ndUhg8ZL, int INDEX0_habLccSPNH) {

	kernel_cuda_wrapper_incr_hVGVGWenmY<<<1, 168>>> (initial_jxAAR4OqHM, result_25ndUhg8ZL, INDEX0_habLccSPNH);
}

__global__ 
void kernel_cuda_wrapper_incr_hVGVGWenmY(int32_t* initial_jxAAR4OqHM, int32_t* result_25ndUhg8ZL, int INDEX0_habLccSPNH) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 0;
	if (tid < 100) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX1_habLccSPNH = tid * exec_range + range_iterator + 100;
		if (tid < 100) {
			INDEX1_habLccSPNH -= 100;
		}
		result_25ndUhg8ZL[(INDEX1_habLccSPNH)] = initial_jxAAR4OqHM[(INDEX1_habLccSPNH)] + 1;
	}
}


