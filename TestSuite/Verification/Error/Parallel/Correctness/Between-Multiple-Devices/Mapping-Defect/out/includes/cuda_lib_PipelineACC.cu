#include "hip/hip_runtime.h"
/**


This file implements the header for the Thread Pool and barrier implementation with PThreads.


*/
#include <stdio.h>
#include <stdlib.h>
#include "cuda_lib_PipelineACC.hxx"
#include "cuda_lib_PipelineACC.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "Patternlib.hxx"




template<typename T>
__global__
void cuda_reduce_sum(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] += sdata[tid + 32];
        if (n >= 32) sdata[tid] += sdata[tid + 16];
        if (n >= 16) sdata[tid] += sdata[tid + 8];
        if (n >= 8) sdata[tid] += sdata[tid + 4];
        if (n >= 4) sdata[tid] += sdata[tid + 2];
        if (n >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_times(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] *= sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] *= sdata[tid + 32];
        if (n >= 32) sdata[tid] *= sdata[tid + 16];
        if (n >= 16) sdata[tid] *= sdata[tid + 8];
        if (n >= 8) sdata[tid] *= sdata[tid + 4];
        if (n >= 4) sdata[tid] *= sdata[tid + 2];
        if (n >= 2) sdata[tid] *= sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_min(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_max(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = max(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = max(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = max(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = max(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = max(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = max(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

void cuda_wrapper_increment_2XlMsn6B8K(int32_t* initial_Pfe9Oio93E, int32_t* intermediate_ZFsDXzWPC8 ) {

	kernel_cuda_wrapper_increment_2XlMsn6B8K<<<1, 168>>> (initial_Pfe9Oio93E, intermediate_ZFsDXzWPC8 );
}

__global__ 
void kernel_cuda_wrapper_increment_2XlMsn6B8K(int32_t*  initial_Pfe9Oio93E, int32_t* intermediate_ZFsDXzWPC8 ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 1;
	if (tid < 32) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_2XlMsn6B8K = tid * exec_range + range_iterator + 32;
		if (tid < 32) {
			INDEX_2XlMsn6B8K -= 32;
		}
		intermediate_ZFsDXzWPC8[(INDEX_2XlMsn6B8K)] = initial_Pfe9Oio93E[(INDEX_2XlMsn6B8K)] + 1;
	}
}

void cuda_wrapper_increment_xxGyRu4JSm(int32_t* intermediate_ZFsDXzWPC8, int32_t* result_ZMrKKHa9pD ) {

	kernel_cuda_wrapper_increment_xxGyRu4JSm<<<1, 168>>> (intermediate_ZFsDXzWPC8, result_ZMrKKHa9pD );
}

__global__ 
void kernel_cuda_wrapper_increment_xxGyRu4JSm(int32_t*  intermediate_ZFsDXzWPC8, int32_t* result_ZMrKKHa9pD ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 1;
	if (tid < 32) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_xxGyRu4JSm = tid * exec_range + range_iterator + 32;
		if (tid < 32) {
			INDEX_xxGyRu4JSm -= 32;
		}
		result_ZMrKKHa9pD[(INDEX_xxGyRu4JSm)] = intermediate_ZFsDXzWPC8[(INDEX_xxGyRu4JSm)] + 1;
	}
}


