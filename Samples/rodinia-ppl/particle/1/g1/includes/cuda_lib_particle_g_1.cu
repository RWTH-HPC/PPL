#include "hip/hip_runtime.h"
/**


This file implements the header for the Thread Pool and barrier implementation with PThreads.


*/
#include <stdio.h>
#include <stdlib.h>
#include "cuda_lib_particle_g_1.hxx"
#include "cuda_lib_particle_g_1.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "Patternlib.hxx"


float powi(float x, int32_t n) {
	float res_5g3ylNY1J1;
	res_5g3ylNY1J1 = 1;
	for ( int32_t i = 0; i < n; i++ ) {
		res_5g3ylNY1J1 *= x;
	}
		return res_5g3ylNY1J1;
}
int32_t roundDouble(double value) {
	int32_t newValue_xULy1ztDPc;
	newValue_xULy1ztDPc = Cast2Int(value);
	if ((value - newValue_xULy1ztDPc < 0.5)) {
				return newValue_xULy1ztDPc;
	} else {
				return newValue_xULy1ztDPc + 1;
	}
}


template<typename T>
__global__
void cuda_reduce_sum(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] += sdata[tid + 32];
        if (n >= 32) sdata[tid] += sdata[tid + 16];
        if (n >= 16) sdata[tid] += sdata[tid + 8];
        if (n >= 8) sdata[tid] += sdata[tid + 4];
        if (n >= 4) sdata[tid] += sdata[tid + 2];
        if (n >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_times(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] *= sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] *= sdata[tid + 32];
        if (n >= 32) sdata[tid] *= sdata[tid + 16];
        if (n >= 16) sdata[tid] *= sdata[tid + 8];
        if (n >= 8) sdata[tid] *= sdata[tid + 4];
        if (n >= 4) sdata[tid] *= sdata[tid + 2];
        if (n >= 2) sdata[tid] *= sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_min(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_max(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = max(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = max(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = max(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = max(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = max(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = max(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

void cuda_wrapper_u_init_zK3zsoJUg7(double* u1_UstTxnfY90_J45bILqR5e, int32_t* Nparticles_Xtno2sLxiz, double* u_uVrNRP6SnA ) {

	kernel_cuda_wrapper_u_init_zK3zsoJUg7<<<10, 64>>> (u1_UstTxnfY90_J45bILqR5e, Nparticles_Xtno2sLxiz, u_uVrNRP6SnA );
}

__global__ 
void kernel_cuda_wrapper_u_init_zK3zsoJUg7(double*  u1_UstTxnfY90_J45bILqR5e, int32_t*  Nparticles_Xtno2sLxiz, double* u_uVrNRP6SnA ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 15;
	if (tid < 400) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_zK3zsoJUg7 = tid * exec_range + range_iterator + 400;
		if (tid < 400) {
			INDEX_zK3zsoJUg7 -= 400;
		}
		u_uVrNRP6SnA[(INDEX_zK3zsoJUg7)] = u1_UstTxnfY90_J45bILqR5e[0] + INDEX_zK3zsoJUg7 / Nparticles_Xtno2sLxiz[0];
	}
}

void cuda_wrapper_u_init_vlOhsxzeZk(double* u1_UstTxnfY90_J45bILqR5e, int32_t* Nparticles_Xtno2sLxiz, double* u_uVrNRP6SnA ) {

	kernel_cuda_wrapper_u_init_vlOhsxzeZk<<<10, 64>>> (u1_UstTxnfY90_J45bILqR5e, Nparticles_Xtno2sLxiz, u_uVrNRP6SnA );
}

__global__ 
void kernel_cuda_wrapper_u_init_vlOhsxzeZk(double*  u1_UstTxnfY90_J45bILqR5e, int32_t*  Nparticles_Xtno2sLxiz, double* u_uVrNRP6SnA ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 15;
	if (tid < 400) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_vlOhsxzeZk = tid * exec_range + range_iterator + 400;
		if (tid < 400) {
			INDEX_vlOhsxzeZk -= 400;
		}
		u_uVrNRP6SnA[(INDEX_vlOhsxzeZk)] = u1_UstTxnfY90_J45bILqR5e[0] + INDEX_vlOhsxzeZk / Nparticles_Xtno2sLxiz[0];
	}
}


