#include "hip/hip_runtime.h"
/**


This file implements the header for the Thread Pool and barrier implementation with PThreads.


*/
#include <stdio.h>
#include <stdlib.h>
#include "cuda_lib_particle_g_1.hxx"
#include "cuda_lib_particle_g_1.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "Patternlib.hxx"


float powi(float x, int32_t n) {
	float res_PuUbIfNc8X;
	res_PuUbIfNc8X = 1;
	for ( int32_t i = 0; i < n; i++ ) {
		res_PuUbIfNc8X *= x;
	}
		return res_PuUbIfNc8X;
}
int32_t roundDouble(double value) {
	int32_t newValue_F3e5Dfu58E;
	newValue_F3e5Dfu58E = Cast2Int(value);
	if ((value - newValue_F3e5Dfu58E < 0.5)) {
				return newValue_F3e5Dfu58E;
	} else {
				return newValue_F3e5Dfu58E + 1;
	}
}


template<typename T>
__global__
void cuda_reduce_sum(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] += sdata[tid + 32];
        if (n >= 32) sdata[tid] += sdata[tid + 16];
        if (n >= 16) sdata[tid] += sdata[tid + 8];
        if (n >= 8) sdata[tid] += sdata[tid + 4];
        if (n >= 4) sdata[tid] += sdata[tid + 2];
        if (n >= 2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_times(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] *= sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] *= sdata[tid + 32];
        if (n >= 32) sdata[tid] *= sdata[tid + 16];
        if (n >= 16) sdata[tid] *= sdata[tid + 8];
        if (n >= 8) sdata[tid] *= sdata[tid + 4];
        if (n >= 4) sdata[tid] *= sdata[tid + 2];
        if (n >= 2) sdata[tid] *= sdata[tid + 1];
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_min(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

template<typename T>
__global__
void cuda_reduce_max(T* input, T* output, int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = input[tid];

    __syncthreads();

    if (n >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (n >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (n >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

    if (tid < 32) {
        if (n >= 64) sdata[tid] = max(sdata[tid], sdata[tid + 32]);
        if (n >= 32) sdata[tid] = max(sdata[tid], sdata[tid + 16]);
        if (n >= 16) sdata[tid] = max(sdata[tid], sdata[tid + 8]);
        if (n >= 8) sdata[tid] = max(sdata[tid], sdata[tid + 4]);
        if (n >= 4) sdata[tid] = max(sdata[tid], sdata[tid + 2]);
        if (n >= 2) sdata[tid] = max(sdata[tid], sdata[tid + 1]);
    }

    if (tid == 0) output[0] = sdata[0];

}

void cuda_wrapper_u_init_zj9EbHdPzb(double* u1_ZZHMvQb2i6_s96P5Y86vI, int32_t* Nparticles_2f7BRjYtYI, double* u_x64gIli0uq ) {

	kernel_cuda_wrapper_u_init_zj9EbHdPzb<<<10, 64>>> (u1_ZZHMvQb2i6_s96P5Y86vI, Nparticles_2f7BRjYtYI, u_x64gIli0uq );
}

__global__ 
void kernel_cuda_wrapper_u_init_zj9EbHdPzb(double*  u1_ZZHMvQb2i6_s96P5Y86vI, int32_t*  Nparticles_2f7BRjYtYI, double* u_x64gIli0uq ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 15;
	if (tid < 400) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_zj9EbHdPzb = tid * exec_range + range_iterator + 400;
		if (tid < 400) {
			INDEX_zj9EbHdPzb -= 400;
		}
		u_x64gIli0uq[(INDEX_zj9EbHdPzb)] = u1_ZZHMvQb2i6_s96P5Y86vI[0] + INDEX_zj9EbHdPzb / Nparticles_2f7BRjYtYI[0];
	}
}

void cuda_wrapper_u_init_U4raAdIpaG(double* u1_ZZHMvQb2i6_s96P5Y86vI, int32_t* Nparticles_2f7BRjYtYI, double* u_x64gIli0uq ) {

	kernel_cuda_wrapper_u_init_U4raAdIpaG<<<10, 64>>> (u1_ZZHMvQb2i6_s96P5Y86vI, Nparticles_2f7BRjYtYI, u_x64gIli0uq );
}

__global__ 
void kernel_cuda_wrapper_u_init_U4raAdIpaG(double*  u1_ZZHMvQb2i6_s96P5Y86vI, int32_t*  Nparticles_2f7BRjYtYI, double* u_x64gIli0uq ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int exec_range = 15;
	if (tid < 400) {
		exec_range++;
	}
	for ( int range_iterator = 0; range_iterator < exec_range + 0; range_iterator++) {
		int INDEX_U4raAdIpaG = tid * exec_range + range_iterator + 400;
		if (tid < 400) {
			INDEX_U4raAdIpaG -= 400;
		}
		u_x64gIli0uq[(INDEX_U4raAdIpaG)] = u1_ZZHMvQb2i6_s96P5Y86vI[0] + INDEX_U4raAdIpaG / Nparticles_2f7BRjYtYI[0];
	}
}


